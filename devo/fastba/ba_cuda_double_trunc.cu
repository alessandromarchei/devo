#include "hip/hip_runtime.h"

#include <torch/extension.h>
#include <vector>
#include <iostream>
#include <cmath>

#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Parallel.h>


#define DECIMAL_PLACES 2


#define GPU_1D_KERNEL_LOOP(i, n) \
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)

#define NUM_THREADS 256
#define NUM_BLOCKS(batch_size) ((batch_size + NUM_THREADS - 1) / NUM_THREADS)


__global__ void round_tensor_kernel_anyshape(double* data, int64_t numel, const double factor) {
    GPU_1D_KERNEL_LOOP(i, numel) {
        double val = data[i];
        val = roundf(val * factor) / factor;
        data[i] = val;
    }
}

void round_tensor(torch::Tensor& tensor, int decimal_places) {
    double factor = powf(10.0f, (double)decimal_places);

    //std::cout << "[round_tensor] Shape: " << tensor.sizes()
              //<< ", Dtype: " << tensor.dtype()
              //<< ", Device: " << tensor.device()
              //<< ", Decimal places: " << decimal_places << std::endl;

    TORCH_CHECK(tensor.is_cuda(), "round_tensor only supports CUDA tensors.");
    TORCH_CHECK(tensor.scalar_type() == torch::kDouble, "round_tensor only supports double32 tensors.");

    torch::Tensor tensor_flat = tensor.view(-1);

    double* data_ptr = tensor_flat.data_ptr<double>();
    int64_t numel = tensor_flat.numel();

    round_tensor_kernel_anyshape<<<NUM_BLOCKS(numel), NUM_THREADS>>>(data_ptr, numel, factor);
    
    hipDeviceSynchronize();

    //std::cout << "[output] Shape: " << tensor.sizes()
              //<< ", Dtype: " << tensor.dtype()
              //<< ", Device: " << tensor.device()
              //<< std::endl;
    //std::cout << "[round_tensor] Done." << std::endl;
}




// --------------------------- SE3 and SO3 math -----------------------------

__device__ void actSO3(const double *q, const double *X, double *Y) {
    double uv[3];
    uv[0] = 2.0 * (q[1]*X[2] - q[2]*X[1]);
    uv[1] = 2.0 * (q[2]*X[0] - q[0]*X[2]);
    uv[2] = 2.0 * (q[0]*X[1] - q[1]*X[0]);

    Y[0] = X[0] + q[3]*uv[0] + (q[1]*uv[2] - q[2]*uv[1]);
    Y[1] = X[1] + q[3]*uv[1] + (q[2]*uv[0] - q[0]*uv[2]);
    Y[2] = X[2] + q[3]*uv[2] + (q[0]*uv[1] - q[1]*uv[0]);
}

__device__ void actSE3(const double *t, const double *q, const double *X, double *Y) {
    actSO3(q, X, Y);
    Y[3] = X[3];
    Y[0] += X[3] * t[0];
    Y[1] += X[3] * t[1];
    Y[2] += X[3] * t[2];
}

__device__ void adjSE3(const double *t, const double *q, const double *X, double *Y) {
    double qinv[4] = {-q[0], -q[1], -q[2], q[3]};
    actSO3(qinv, &X[0], &Y[0]);
    actSO3(qinv, &X[3], &Y[3]);

    double u[3], v[3];
    u[0] = t[2]*X[1] - t[1]*X[2];
    u[1] = t[0]*X[2] - t[2]*X[0];
    u[2] = t[1]*X[0] - t[0]*X[1];

    actSO3(qinv, u, v);
    Y[3] += v[0];
    Y[4] += v[1];
    Y[5] += v[2];
}

__device__ void relSE3(const double *ti, const double *qi, const double *tj, const double *qj, double *tij, double *qij) {
    qij[0] = -qj[3] * qi[0] + qj[0] * qi[3] - qj[1] * qi[2] + qj[2] * qi[1];
    qij[1] = -qj[3] * qi[1] + qj[1] * qi[3] - qj[2] * qi[0] + qj[0] * qi[2];
    qij[2] = -qj[3] * qi[2] + qj[2] * qi[3] - qj[0] * qi[1] + qj[1] * qi[0];
    qij[3] =  qj[3] * qi[3] + qj[0] * qi[0] + qj[1] * qi[1] + qj[2] * qi[2];

    actSO3(qij, ti, tij);
    tij[0] = tj[0] - tij[0];
    tij[1] = tj[1] - tij[1];
    tij[2] = tj[2] - tij[2];
}


__device__ void
crossInplace(const double* a, double *b) {
  double x[3] = {
    a[1]*b[2] - a[2]*b[1],
    a[2]*b[0] - a[0]*b[2],
    a[0]*b[1] - a[1]*b[0], 
  };

  b[0] = x[0];
  b[1] = x[1];
  b[2] = x[2];
}


  
__device__ void
expSO3(const double *phi, double* q) {
  // SO3 exponential map
  double theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  double theta_p4 = theta_sq * theta_sq;

  double theta = sqrtf(theta_sq);
  double imag, real;

  if (theta_sq < 1e-8) {
    imag = 0.5 - (1.0/48.0)*theta_sq + (1.0/3840.0)*theta_p4;
    real = 1.0 - (1.0/ 8.0)*theta_sq + (1.0/ 384.0)*theta_p4;
  } else {
    imag = sinf(0.5 * theta) / theta;
    real = cosf(0.5 * theta);
  }

  q[0] = imag * phi[0];
  q[1] = imag * phi[1];
  q[2] = imag * phi[2];
  q[3] = real;

}


__device__ void
expSE3(const double *xi, double* t, double* q) {
  // SE3 exponential map

  expSO3(xi + 3, q);
  double tau[3] = {xi[0], xi[1], xi[2]};
  double phi[3] = {xi[3], xi[4], xi[5]};

  double theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  double theta = sqrtf(theta_sq);

  t[0] = tau[0]; 
  t[1] = tau[1]; 
  t[2] = tau[2];

  if (theta > 1e-4) {
    double a = (1 - cosf(theta)) / theta_sq;
    crossInplace(phi, tau);
    t[0] += a * tau[0];
    t[1] += a * tau[1];
    t[2] += a * tau[2];

    double b = (theta - sinf(theta)) / (theta * theta_sq);
    crossInplace(phi, tau);
    t[0] += b * tau[0];
    t[1] += b * tau[1];
    t[2] += b * tau[2];
  }
}




__device__ void
retrSE3(const double *xi, const double* t, const double* q, double* t1, double* q1) {
  // retraction on SE3 manifold

  double dt[3] = {0, 0, 0};
  double dq[4] = {0, 0, 0, 1};
  
  expSE3(xi, dt, dq);

  q1[0] = dq[3] * q[0] + dq[0] * q[3] + dq[1] * q[2] - dq[2] * q[1];
  q1[1] = dq[3] * q[1] + dq[1] * q[3] + dq[2] * q[0] - dq[0] * q[2];
  q1[2] = dq[3] * q[2] + dq[2] * q[3] + dq[0] * q[1] - dq[1] * q[0];
  q1[3] = dq[3] * q[3] - dq[0] * q[0] - dq[1] * q[1] - dq[2] * q[2];

  actSO3(dq, t, t1);
  t1[0] += dt[0];
  t1[1] += dt[1];
  t1[2] += dt[2];
}





__global__ void pose_retr_kernel(const int t0, const int t1,
    torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> poses,
    torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> update)
{
  GPU_1D_KERNEL_LOOP(i, t1 - t0) {
    const double t = t0 + i;
    double t1[3], t0[3] = { poses[t][0], poses[t][1], poses[t][2] };
    double q1[4], q0[4] = { poses[t][3], poses[t][4], poses[t][5], poses[t][6] };

    double xi[6] = {
      update[i][0],
      update[i][1],
      update[i][2],
      update[i][3],
      update[i][4],
      update[i][5],
    };

    retrSE3(xi, t0, q0, t1, q1);

    poses[t][0] = t1[0];
    poses[t][1] = t1[1];
    poses[t][2] = t1[2];
    poses[t][3] = q1[0];
    poses[t][4] = q1[1];
    poses[t][5] = q1[2];
    poses[t][6] = q1[3];
  }
}


__global__ void patch_retr_kernel(
    torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> index,
    torch::PackedTensorAccessor32<double,4,torch::RestrictPtrTraits> patches,
    torch::PackedTensorAccessor32<double,1,torch::RestrictPtrTraits> update)
{
  GPU_1D_KERNEL_LOOP(n, index.size(0)) {
    const int p = patches.size(2);
    const int ix = index[n];
  
    double d = patches[ix][2][0][0];
    d = d + update[n];
    d = (d > 20) ? 1.0 : d;
    d = max(d, 1e-4);

    for (int i=0; i<p; i++) {
      for (int j=0; j<p; j++) {
        patches[ix][2][i][j] = d;
      }
    }
  }
}




__global__ void reprojection_residuals_and_hessian(
    const torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<double,4,torch::RestrictPtrTraits> patches,
    const torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> target,
    const torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> weight,
    const torch::PackedTensorAccessor32<double,1,torch::RestrictPtrTraits> lmbda,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> kk,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ku,
    torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> B,
    torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> E,
    torch::PackedTensorAccessor32<double,1,torch::RestrictPtrTraits> C,
    torch::PackedTensorAccessor32<double,1,torch::RestrictPtrTraits> v,
    torch::PackedTensorAccessor32<double,1,torch::RestrictPtrTraits> u, const int t0)
{

  __shared__ double fx, fy, cx, cy;
  if (threadIdx.x == 0) {
    fx = intrinsics[0][0];
    fy = intrinsics[0][1];
    cx = intrinsics[0][2];
    cy = intrinsics[0][3];
  }

  __syncthreads();

  GPU_1D_KERNEL_LOOP(n, ii.size(0)) {
    int k = ku[n];
    int ix = ii[n];
    int jx = jj[n];
    int kx = kk[n];

    double ti[3] = { poses[ix][0], poses[ix][1], poses[ix][2] };
    double tj[3] = { poses[jx][0], poses[jx][1], poses[jx][2] };
    double qi[4] = { poses[ix][3], poses[ix][4], poses[ix][5], poses[ix][6] };
    double qj[4] = { poses[jx][3], poses[jx][4], poses[jx][5], poses[jx][6] };

    double Xi[4], Xj[4];
    Xi[0] = (patches[kx][0][1][1] - cx) / fx;
    Xi[1] = (patches[kx][1][1][1] - cy) / fy;
    Xi[2] = 1.0;
    Xi[3] = patches[kx][2][1][1];
    
    double tij[3], qij[4];
    relSE3(ti, qi, tj, qj, tij, qij);
    actSE3(tij, qij, Xi, Xj);

    const double X = Xj[0];
    const double Y = Xj[1];
    const double Z = Xj[2];
    const double W = Xj[3];

    const double d = (Z >= 0.2) ? 1.0 / Z : 0.0; 
    const double d2 = d * d;

    const double x1 = fx * (X / Z) + cx;
    const double y1 = fy * (Y / Z) + cy;

    const double rx = target[n][0] - x1;
    const double ry = target[n][1] - y1;

    const bool in_bounds = (sqrt(rx*rx + ry*ry) < 128) && (Z > 0.2) &&
      (x1 > -64) && (y1 > -64) && (x1 < 2*cx + 64) && (y1 < 2*cy + 64);

    const double mask = in_bounds ? 1.0 : 0.0;

    ix = ix - t0;
    jx = jx - t0;

    {
      const double r = target[n][0] - x1;
      const double w = mask * weight[n][0];

      double Jz = fx * (tij[0] * d - tij[2] * (X * d2));
      double Ji[6], Jj[6] = {fx*W*d, 0, fx*-X*W*d2, fx*-X*Y*d2, fx*(1+X*X*d2), fx*-Y*d};

      adjSE3(tij, qij, Jj, Ji);

      for (int i=0; i<6; i++) {
        for (int j=0; j<6; j++) {
          if (ix >= 0)
            atomicAdd(&B[6*ix+i][6*ix+j],  w * Ji[i] * Ji[j]);
          if (jx >= 0)
            atomicAdd(&B[6*jx+i][6*jx+j],  w * Jj[i] * Jj[j]);
          if (ix >= 0 && jx >= 0) {
            atomicAdd(&B[6*ix+i][6*jx+j], -w * Ji[i] * Jj[j]);
            atomicAdd(&B[6*jx+i][6*ix+j], -w * Jj[i] * Ji[j]);
          }
        }
      }

      for (int i=0; i<6; i++) {
        if (ix >= 0)
          atomicAdd(&E[6*ix+i][k], -w * Jz * Ji[i]);
        if (jx >= 0)
          atomicAdd(&E[6*jx+i][k],  w * Jz * Jj[i]);
      }

      for (int i=0; i<6; i++) {
        if (ix >= 0)
          atomicAdd(&v[6*ix+i], -w * r * Ji[i]);
        if (jx >= 0)
          atomicAdd(&v[6*jx+i],  w * r * Jj[i]);
      }

      atomicAdd(&C[k], w * Jz * Jz);
      atomicAdd(&u[k], w *  r * Jz);
    }

    {
      const double r = target[n][1] - y1;
      const double w = mask * weight[n][1];
      
      double Jz = fy * (tij[1] * d - tij[2] * (Y * d2));
      double Ji[6], Jj[6] = {0, fy*W*d, fy*-Y*W*d2, fy*(-1-Y*Y*d2), fy*(X*Y*d2), fy*X*d};
      
      adjSE3(tij, qij, Jj, Ji);

      for (int i=0; i<6; i++) {
        for (int j=0; j<6; j++) {
          if (ix >= 0)
            atomicAdd(&B[6*ix+i][6*ix+j],  w * Ji[i] * Ji[j]);
          if (jx >= 0)
            atomicAdd(&B[6*jx+i][6*jx+j],  w * Jj[i] * Jj[j]);
          if (ix >= 0 && jx >= 0) {
            atomicAdd(&B[6*ix+i][6*jx+j], -w * Ji[i] * Jj[j]);
            atomicAdd(&B[6*jx+i][6*ix+j], -w * Jj[i] * Ji[j]);
          }
        }
      }

      for (int i=0; i<6; i++) {
        if (ix >= 0)
          atomicAdd(&E[6*ix+i][k], -w * Jz * Ji[i]);
        if (jx >= 0)
          atomicAdd(&E[6*jx+i][k],  w * Jz * Jj[i]);
      }

      for (int i=0; i<6; i++) {
        if (ix >= 0)
          atomicAdd(&v[6*ix+i], -w * r * Ji[i]);
        if (jx >= 0)
          atomicAdd(&v[6*jx+i],  w * r * Jj[i]);
      }

      atomicAdd(&C[k], w * Jz * Jz);
      atomicAdd(&u[k], w *  r * Jz);
    }
  }
}



__global__ void reproject(
    const torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<double,4,torch::RestrictPtrTraits> patches,
    const torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> kk,
    torch::PackedTensorAccessor32<double,4,torch::RestrictPtrTraits> coords) {

  __shared__ double fx, fy, cx, cy;
  if (threadIdx.x == 0) {
    fx = intrinsics[0][0];
    fy = intrinsics[0][1];
    cx = intrinsics[0][2];
    cy = intrinsics[0][3];
  }

  __syncthreads();

  GPU_1D_KERNEL_LOOP(n, ii.size(0)) {
    int ix = ii[n];
    int jx = jj[n];
    int kx = kk[n];

    double ti[3] = { poses[ix][0], poses[ix][1], poses[ix][2] };
    double tj[3] = { poses[jx][0], poses[jx][1], poses[jx][2] };
    double qi[4] = { poses[ix][3], poses[ix][4], poses[ix][5], poses[ix][6] };
    double qj[4] = { poses[jx][3], poses[jx][4], poses[jx][5], poses[jx][6] };

    double tij[3], qij[4];
    relSE3(ti, qi, tj, qj, tij, qij);

    double Xi[4], Xj[4];
    for (int i=0; i<patches.size(2); i++) {
      for (int j=0; j<patches.size(3); j++) {
        
        Xi[0] = (patches[kx][0][i][j] - cx) / fx;
        Xi[1] = (patches[kx][1][i][j] - cy) / fy;
        Xi[2] = 1.0;
        Xi[3] = patches[kx][2][i][j];

        actSE3(tij, qij, Xi, Xj);

        coords[n][0][i][j] = fx * (Xj[0] / Xj[2]) + cx;
        coords[n][1][i][j] = fy * (Xj[1] / Xj[2]) + cy;
        // coords[n][2][i][j] = 1.0 / Xj[2];

      }
    }
  }
}




torch::Tensor cuda_reproject(
    torch::Tensor poses,
    torch::Tensor patches,
    torch::Tensor intrinsics,
    torch::Tensor ii,
    torch::Tensor jj, 
    torch::Tensor kk)
{

  const int N = ii.size(0);
  const int P = patches.size(3); // patch size

  poses = poses.view({-1, 7});
  patches = patches.view({-1,3,P,P});
  intrinsics = intrinsics.view({-1, 4});

  auto opts = torch::TensorOptions()
    .dtype(torch::kDouble).device(torch::kCUDA);

  torch::Tensor coords = torch::empty({N, 2, P, P}, opts);

  reproject<<<NUM_BLOCKS(N), NUM_THREADS>>>(
    poses.packed_accessor32<double,2,torch::RestrictPtrTraits>(),
    patches.packed_accessor32<double,4,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<double,2,torch::RestrictPtrTraits>(),
    ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    kk.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    coords.packed_accessor32<double,4,torch::RestrictPtrTraits>());

  return coords.view({1, N, 2, P, P});

}


// ---------------------- Bundle Adjustment Core -----------------------------

std::vector<torch::Tensor> cuda_ba(
    torch::Tensor poses,
    torch::Tensor patches,
    torch::Tensor intrinsics,
    torch::Tensor target,
    torch::Tensor weight,
    torch::Tensor lmbda,
    torch::Tensor ii,
    torch::Tensor jj,
    torch::Tensor kk,
    const int t0, const int t1, const int iterations,
  const int decimal_places = DECIMAL_PLACES)
{
    auto ktuple = torch::_unique(kk, true, true);
    torch::Tensor kx = std::get<0>(ktuple);
    torch::Tensor ku = std::get<1>(ktuple);

    const int N = t1 - t0;
    const int M = kx.size(0);
    const int P = patches.size(3);

    auto opts = torch::TensorOptions().dtype(torch::kDouble).device(torch::kCUDA);

    poses = poses.view({-1, 7});
    patches = patches.view({-1, 3, P, P});
    intrinsics = intrinsics.view({-1, 4});
    target = target.view({-1, 2});
    weight = weight.view({-1, 2});

    torch::Tensor B = torch::empty({6 * N, 6 * N}, opts);
    torch::Tensor E = torch::empty({6 * N, 1 * M}, opts);
    torch::Tensor C = torch::empty({M}, opts);
    torch::Tensor v = torch::empty({6 * N}, opts);
    torch::Tensor u = torch::empty({1 * M}, opts);


    for (int itr = 0; itr < iterations; itr++) {
        B.zero_();
        E.zero_();
        C.zero_();
        v.zero_();
        u.zero_();
        v = v.view({6*N});
        u = u.view({1*M});
        reprojection_residuals_and_hessian<<<NUM_BLOCKS(ii.size(0)), NUM_THREADS>>>(
            poses.packed_accessor32<double, 2, torch::RestrictPtrTraits>(),
            patches.packed_accessor32<double, 4, torch::RestrictPtrTraits>(),
            intrinsics.packed_accessor32<double, 2, torch::RestrictPtrTraits>(),
            target.packed_accessor32<double, 2, torch::RestrictPtrTraits>(),
            weight.packed_accessor32<double, 2, torch::RestrictPtrTraits>(),
            lmbda.packed_accessor32<double, 1, torch::RestrictPtrTraits>(),
            ii.packed_accessor32<long, 1, torch::RestrictPtrTraits>(),
            jj.packed_accessor32<long, 1, torch::RestrictPtrTraits>(),
            kk.packed_accessor32<long, 1, torch::RestrictPtrTraits>(),
            ku.packed_accessor32<long, 1, torch::RestrictPtrTraits>(),
            B.packed_accessor32<double, 2, torch::RestrictPtrTraits>(),
            E.packed_accessor32<double, 2, torch::RestrictPtrTraits>(),
            C.packed_accessor32<double, 1, torch::RestrictPtrTraits>(),
            v.packed_accessor32<double, 1, torch::RestrictPtrTraits>(),
            u.packed_accessor32<double, 1, torch::RestrictPtrTraits>(),
            t0);
        
        v = v.view({6*N, 1});
        u = u.view({1*M, 1});
        //std::cout << "truncating B" << std::endl;
        round_tensor(B, decimal_places);

        //std::cout << "truncating E" << std::endl;
        round_tensor(E, decimal_places);

        //std::cout << "truncating C" << std::endl;
        round_tensor(C, decimal_places);

        //std::cout << "truncating v" << std::endl;
        round_tensor(v, decimal_places);

        //std::cout << "truncating u" << std::endl;
        round_tensor(u, decimal_places);

        torch::Tensor Q = 1.0 / (C + lmbda).view({1, M});

        //std::cout << "truncating Q" << std::endl;
        round_tensor(Q, decimal_places);

        if (t1 - t0 == 0) {
            torch::Tensor Qt = Q.transpose(0, 1);
            torch::Tensor dZ = Qt * u;
            dZ = dZ.view({M});
            
            //std::cout << "truncating dZ" << std::endl;
            round_tensor(dZ, decimal_places);

            patch_retr_kernel<<<NUM_BLOCKS(M), NUM_THREADS>>>(
                kx.packed_accessor32<long, 1, torch::RestrictPtrTraits>(),
                patches.packed_accessor32<double, 4, torch::RestrictPtrTraits>(),
                dZ.packed_accessor32<double, 1, torch::RestrictPtrTraits>());
        }
        else {
            torch::Tensor EQ = E * Q;
            
            torch::Tensor Et = E.transpose(0, 1);
            torch::Tensor Qt = Q.transpose(0, 1);

            torch::Tensor S = B - torch::matmul(EQ, Et);
            torch::Tensor y = v - torch::matmul(EQ, u);

            //std::cout << "truncating S" << std::endl;
            round_tensor(S, decimal_places);

            //std::cout << "truncating y" << std::endl;
            round_tensor(y, decimal_places);

            torch::Tensor I = torch::eye(6 * N, opts);
            S += I * (1e-4 * S + 1.0);

            //std::cout << "torch linalg cholesky" << std::endl;
            torch::Tensor U = torch::linalg::cholesky(S);

            //std::cout << "torch linalg cholesky_solve" << std::endl;
            torch::Tensor dX = torch::cholesky_solve(y, U);

            //std::cout << "matmul" << std::endl;  
            torch::Tensor dZ = Qt * (u - torch::matmul(Et, dX));

            dX = dX.view({N, 6});
            dZ = dZ.view({M});

            //std::cout << "truncating dX" << std::endl;
            round_tensor(dX, decimal_places);

            //std::cout << "truncating dZ" << std::endl;
            round_tensor(dZ, decimal_places);

            pose_retr_kernel<<<NUM_BLOCKS(N), NUM_THREADS>>>(
                t0, t1,
                poses.packed_accessor32<double, 2, torch::RestrictPtrTraits>(),
                dX.packed_accessor32<double, 2, torch::RestrictPtrTraits>());

            patch_retr_kernel<<<NUM_BLOCKS(M), NUM_THREADS>>>(
                kx.packed_accessor32<long, 1, torch::RestrictPtrTraits>(),
                patches.packed_accessor32<double, 4, torch::RestrictPtrTraits>(),
                dZ.packed_accessor32<double, 1, torch::RestrictPtrTraits>());
        }
    }

    return {};
}
