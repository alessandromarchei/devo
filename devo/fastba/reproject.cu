#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <iostream>

#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Parallel.h>


#define GPU_1D_KERNEL_LOOP(i, n) \
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i<n; i += blockDim.x * gridDim.x)


#define NUM_THREADS 256
#define NUM_BLOCKS(batch_size) ((batch_size + NUM_THREADS - 1) / NUM_THREADS)


__device__ void
actSO3_gpu(const float *q, const float *X, float *Y) {
  float uv[3];
  uv[0] = 2.0 * (q[1]*X[2] - q[2]*X[1]);
  uv[1] = 2.0 * (q[2]*X[0] - q[0]*X[2]);
  uv[2] = 2.0 * (q[0]*X[1] - q[1]*X[0]);

  Y[0] = X[0] + q[3]*uv[0] + (q[1]*uv[2] - q[2]*uv[1]);
  Y[1] = X[1] + q[3]*uv[1] + (q[2]*uv[0] - q[0]*uv[2]);
  Y[2] = X[2] + q[3]*uv[2] + (q[0]*uv[1] - q[1]*uv[0]);
}

__device__  void
actSE3_gpu(const float *t, const float *q, const float *X, float *Y) {
  actSO3_gpu(q, X, Y);
  Y[3] = X[3];
  Y[0] += X[3] * t[0];
  Y[1] += X[3] * t[1];
  Y[2] += X[3] * t[2];
}

__device__ void
adjSE3_gpu(const float *t, const float *q, const float *X, float *Y) {
  float qinv[4] = {-q[0], -q[1], -q[2], q[3]};
  actSO3_gpu(qinv, &X[0], &Y[0]);
  actSO3_gpu(qinv, &X[3], &Y[3]);

  float u[3], v[3];
  u[0] = t[2]*X[1] - t[1]*X[2];
  u[1] = t[0]*X[2] - t[2]*X[0];
  u[2] = t[1]*X[0] - t[0]*X[1];

  actSO3_gpu(qinv, u, v);
  Y[3] += v[0];
  Y[4] += v[1];
  Y[5] += v[2];
}

__device__ void 
relSE3_gpu(const float *ti, const float *qi, const float *tj, const float *qj, float *tij, float *qij) {
  qij[0] = -qj[3] * qi[0] + qj[0] * qi[3] - qj[1] * qi[2] + qj[2] * qi[1],
  qij[1] = -qj[3] * qi[1] + qj[1] * qi[3] - qj[2] * qi[0] + qj[0] * qi[2],
  qij[2] = -qj[3] * qi[2] + qj[2] * qi[3] - qj[0] * qi[1] + qj[1] * qi[0],
  qij[3] =  qj[3] * qi[3] + qj[0] * qi[0] + qj[1] * qi[1] + qj[2] * qi[2],

  actSO3_gpu(qij, ti, tij);
  tij[0] = tj[0] - tij[0];
  tij[1] = tj[1] - tij[1];
  tij[2] = tj[2] - tij[2];
}

  
__device__ void
expSO3_gpu(const float *phi, float* q) {
  // SO3 exponential map
  float theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  float theta_p4 = theta_sq * theta_sq;

  float theta = sqrtf(theta_sq);
  float imag, real;

  if (theta_sq < 1e-8) {
    imag = 0.5 - (1.0/48.0)*theta_sq + (1.0/3840.0)*theta_p4;
    real = 1.0 - (1.0/ 8.0)*theta_sq + (1.0/ 384.0)*theta_p4;
  } else {
    imag = sinf(0.5 * theta) / theta;
    real = cosf(0.5 * theta);
  }

  q[0] = imag * phi[0];
  q[1] = imag * phi[1];
  q[2] = imag * phi[2];
  q[3] = real;

}

__device__ void
crossInplace_gpu(const float* a, float *b) {
  float x[3] = {
    a[1]*b[2] - a[2]*b[1],
    a[2]*b[0] - a[0]*b[2],
    a[0]*b[1] - a[1]*b[0], 
  };

  b[0] = x[0];
  b[1] = x[1];
  b[2] = x[2];
}

__device__ void
expSE3_gpu(const float *xi, float* t, float* q) {
  // SE3 exponential map

  expSO3_gpu(xi + 3, q);
  float tau[3] = {xi[0], xi[1], xi[2]};
  float phi[3] = {xi[3], xi[4], xi[5]};

  float theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  float theta = sqrtf(theta_sq);

  t[0] = tau[0]; 
  t[1] = tau[1]; 
  t[2] = tau[2];

  if (theta > 1e-4) {
    float a = (1 - cosf(theta)) / theta_sq;
    crossInplace_gpu(phi, tau);
    t[0] += a * tau[0];
    t[1] += a * tau[1];
    t[2] += a * tau[2];

    float b = (theta - sinf(theta)) / (theta * theta_sq);
    crossInplace_gpu(phi, tau);
    t[0] += b * tau[0];
    t[1] += b * tau[1];
    t[2] += b * tau[2];
  }
}


__device__ void
retrSE3_gpu(const float *xi, const float* t, const float* q, float* t1, float* q1) {
  // retraction on SE3 manifold

  float dt[3] = {0, 0, 0};
  float dq[4] = {0, 0, 0, 1};
  
  expSE3_gpu(xi, dt, dq);

  q1[0] = dq[3] * q[0] + dq[0] * q[3] + dq[1] * q[2] - dq[2] * q[1];
  q1[1] = dq[3] * q[1] + dq[1] * q[3] + dq[2] * q[0] - dq[0] * q[2];
  q1[2] = dq[3] * q[2] + dq[2] * q[3] + dq[0] * q[1] - dq[1] * q[0];
  q1[3] = dq[3] * q[3] - dq[0] * q[0] - dq[1] * q[1] - dq[2] * q[2];

  actSO3_gpu(dq, t, t1);
  t1[0] += dt[0];
  t1[1] += dt[1];
  t1[2] += dt[2];
}




__global__ void reproject(
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> patches,
    const torch::PackedTensorAccessor32<float,2,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> kk,
    torch::PackedTensorAccessor32<float,4,torch::RestrictPtrTraits> coords) {

  __shared__ float fx, fy, cx, cy;
  if (threadIdx.x == 0) {
    fx = intrinsics[0][0];
    fy = intrinsics[0][1];
    cx = intrinsics[0][2];
    cy = intrinsics[0][3];
  }

  __syncthreads();

  GPU_1D_KERNEL_LOOP(n, ii.size(0)) {
    int ix = ii[n];
    int jx = jj[n];
    int kx = kk[n];

    float ti[3] = { poses[ix][0], poses[ix][1], poses[ix][2] };
    float tj[3] = { poses[jx][0], poses[jx][1], poses[jx][2] };
    float qi[4] = { poses[ix][3], poses[ix][4], poses[ix][5], poses[ix][6] };
    float qj[4] = { poses[jx][3], poses[jx][4], poses[jx][5], poses[jx][6] };

    float tij[3], qij[4];
    relSE3_gpu(ti, qi, tj, qj, tij, qij);

    float Xi[4], Xj[4];
    for (int i=0; i<patches.size(2); i++) {
      for (int j=0; j<patches.size(3); j++) {
        
        Xi[0] = (patches[kx][0][i][j] - cx) / fx;
        Xi[1] = (patches[kx][1][i][j] - cy) / fy;
        Xi[2] = 1.0;
        Xi[3] = patches[kx][2][i][j];

        actSE3_gpu(tij, qij, Xi, Xj);

        coords[n][0][i][j] = fx * (Xj[0] / Xj[2]) + cx;
        coords[n][1][i][j] = fy * (Xj[1] / Xj[2]) + cy;
        // coords[n][2][i][j] = 1.0 / Xj[2];

      }
    }
  }
}



torch::Tensor cuda_reproject(
    torch::Tensor poses,
    torch::Tensor patches,
    torch::Tensor intrinsics,
    torch::Tensor ii,
    torch::Tensor jj, 
    torch::Tensor kk)
{

  const int N = ii.size(0);
  const int P = patches.size(3); // patch size

  poses = poses.view({-1, 7});
  patches = patches.view({-1,3,P,P});
  intrinsics = intrinsics.view({-1, 4});

  auto opts = torch::TensorOptions()
    .dtype(torch::kFloat32).device(torch::kCUDA);

  torch::Tensor coords = torch::empty({N, 2, P, P}, opts);

  reproject<<<NUM_BLOCKS(N), NUM_THREADS>>>(
    poses.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    patches.packed_accessor32<float,4,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<float,2,torch::RestrictPtrTraits>(),
    ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    kk.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    coords.packed_accessor32<float,4,torch::RestrictPtrTraits>());

  return coords.view({1, N, 2, P, P});

}