#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <vector>
#include <iostream>


#include <ATen/ATen.h>
#include <ATen/NativeFunctions.h>
#include <ATen/Parallel.h>


#define GPU_1D_KERNEL_LOOP(i, n) \
  for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i<n; i += blockDim.x * gridDim.x)


#define NUM_THREADS 256
#define NUM_BLOCKS(batch_size) ((batch_size + NUM_THREADS - 1) / NUM_THREADS)

void log_tensor_shape(const torch::Tensor& t, const std::string& name) {
    //printf("[LOG] Tensor %s shape: ", name.c_str());
    for (const auto& s : t.sizes()) std::cout << s << " ";
    std::cout << std::endl;
}

template <typename T, size_t N>
void log_accessor_shape(const at::TensorAccessor<T, N>& accessor, const std::string& name) {
    //printf("[LOG] Tensor %s shape: ", name.c_str());
    for (size_t i = 0; i < N; ++i) {
        std::cout << accessor.size(i) << " ";
    }
    std::cout << std::endl;
}

void log_tensor_stats(const torch::Tensor& t, const std::string& name) {
    auto t_cpu = t.detach().cpu();
    std::cout << "---- " << name << " ----" << std::endl;
    std::cout << "Shape: [";
    for (const auto& s : t_cpu.sizes()) {
        std::cout << s << " ";
    }
    std::cout << "]" << std::endl;

    std::cout << "Dtype: " << t_cpu.dtype() << std::endl;

    std::cout << "Mean: " << t_cpu.mean().item<double>() << std::endl;
    std::cout << "Std: " << t_cpu.std().item<double>() << std::endl;
    std::cout << "Min: " << t_cpu.min().item<double>() << std::endl;
    std::cout << "Max: " << t_cpu.max().item<double>() << std::endl;
    std::cout << "--------------------" << std::endl;
}


__device__ void
actSO3(const double *q, const double *X, double *Y) {
  double uv[3];
  uv[0] = 2.0 * (q[1]*X[2] - q[2]*X[1]);
  uv[1] = 2.0 * (q[2]*X[0] - q[0]*X[2]);
  uv[2] = 2.0 * (q[0]*X[1] - q[1]*X[0]);

  Y[0] = X[0] + q[3]*uv[0] + (q[1]*uv[2] - q[2]*uv[1]);
  Y[1] = X[1] + q[3]*uv[1] + (q[2]*uv[0] - q[0]*uv[2]);
  Y[2] = X[2] + q[3]*uv[2] + (q[0]*uv[1] - q[1]*uv[0]);
}

__device__  void
actSE3(const double *t, const double *q, const double *X, double *Y) {
  actSO3(q, X, Y);
  Y[3] = X[3];
  Y[0] += X[3] * t[0];
  Y[1] += X[3] * t[1];
  Y[2] += X[3] * t[2];
}

__device__ void
adjSE3(const double *t, const double *q, const double *X, double *Y) {
  double qinv[4] = {-q[0], -q[1], -q[2], q[3]};
  actSO3(qinv, &X[0], &Y[0]);
  actSO3(qinv, &X[3], &Y[3]);

  double u[3], v[3];
  u[0] = t[2]*X[1] - t[1]*X[2];
  u[1] = t[0]*X[2] - t[2]*X[0];
  u[2] = t[1]*X[0] - t[0]*X[1];

  actSO3(qinv, u, v);
  Y[3] += v[0];
  Y[4] += v[1];
  Y[5] += v[2];
}

__device__ void 
relSE3(const double *ti, const double *qi, const double *tj, const double *qj, double *tij, double *qij) {
  qij[0] = -qj[3] * qi[0] + qj[0] * qi[3] - qj[1] * qi[2] + qj[2] * qi[1],
  qij[1] = -qj[3] * qi[1] + qj[1] * qi[3] - qj[2] * qi[0] + qj[0] * qi[2],
  qij[2] = -qj[3] * qi[2] + qj[2] * qi[3] - qj[0] * qi[1] + qj[1] * qi[0],
  qij[3] =  qj[3] * qi[3] + qj[0] * qi[0] + qj[1] * qi[1] + qj[2] * qi[2],

  actSO3(qij, ti, tij);
  tij[0] = tj[0] - tij[0];
  tij[1] = tj[1] - tij[1];
  tij[2] = tj[2] - tij[2];
}

  
__device__ void
expSO3(const double *phi, double* q) {
  // SO3 exponential map
  double theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  double theta_p4 = theta_sq * theta_sq;

  double theta = sqrtf(theta_sq);
  double imag, real;

  if (theta_sq < 1e-8) {
    imag = 0.5 - (1.0/48.0)*theta_sq + (1.0/3840.0)*theta_p4;
    real = 1.0 - (1.0/ 8.0)*theta_sq + (1.0/ 384.0)*theta_p4;
  } else {
    imag = sinf(0.5 * theta) / theta;
    real = cosf(0.5 * theta);
  }

  q[0] = imag * phi[0];
  q[1] = imag * phi[1];
  q[2] = imag * phi[2];
  q[3] = real;

}

__device__ void
crossInplace(const double* a, double *b) {
  double x[3] = {
    a[1]*b[2] - a[2]*b[1],
    a[2]*b[0] - a[0]*b[2],
    a[0]*b[1] - a[1]*b[0], 
  };

  b[0] = x[0];
  b[1] = x[1];
  b[2] = x[2];
}

__device__ void
expSE3(const double *xi, double* t, double* q) {
  // SE3 exponential map

  expSO3(xi + 3, q);
  double tau[3] = {xi[0], xi[1], xi[2]};
  double phi[3] = {xi[3], xi[4], xi[5]};

  double theta_sq = phi[0]*phi[0] + phi[1]*phi[1] + phi[2]*phi[2];
  double theta = sqrtf(theta_sq);

  t[0] = tau[0]; 
  t[1] = tau[1]; 
  t[2] = tau[2];

  if (theta > 1e-4) {
    double a = (1 - cosf(theta)) / theta_sq;
    crossInplace(phi, tau);
    t[0] += a * tau[0];
    t[1] += a * tau[1];
    t[2] += a * tau[2];

    double b = (theta - sinf(theta)) / (theta * theta_sq);
    crossInplace(phi, tau);
    t[0] += b * tau[0];
    t[1] += b * tau[1];
    t[2] += b * tau[2];
  }
}


__device__ void
retrSE3(const double *xi, const double* t, const double* q, double* t1, double* q1) {
  // retraction on SE3 manifold

  double dt[3] = {0, 0, 0};
  double dq[4] = {0, 0, 0, 1};
  
  expSE3(xi, dt, dq);

  q1[0] = dq[3] * q[0] + dq[0] * q[3] + dq[1] * q[2] - dq[2] * q[1];
  q1[1] = dq[3] * q[1] + dq[1] * q[3] + dq[2] * q[0] - dq[0] * q[2];
  q1[2] = dq[3] * q[2] + dq[2] * q[3] + dq[0] * q[1] - dq[1] * q[0];
  q1[3] = dq[3] * q[3] - dq[0] * q[0] - dq[1] * q[1] - dq[2] * q[2];

  actSO3(dq, t, t1);
  t1[0] += dt[0];
  t1[1] += dt[1];
  t1[2] += dt[2];
}




__global__ void reproject(
    const torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<double,4,torch::RestrictPtrTraits> patches,
    const torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> kk,
    torch::PackedTensorAccessor32<double,4,torch::RestrictPtrTraits> coords) {

  __shared__ double fx, fy, cx, cy;
  if (threadIdx.x == 0) {
    fx = intrinsics[0][0];
    fy = intrinsics[0][1];
    cx = intrinsics[0][2];
    cy = intrinsics[0][3];
  }

  __syncthreads();

  GPU_1D_KERNEL_LOOP(n, ii.size(0)) {
    int ix = ii[n];
    int jx = jj[n];
    int kx = kk[n];

    double ti[3] = { poses[ix][0], poses[ix][1], poses[ix][2] };
    double tj[3] = { poses[jx][0], poses[jx][1], poses[jx][2] };
    double qi[4] = { poses[ix][3], poses[ix][4], poses[ix][5], poses[ix][6] };
    double qj[4] = { poses[jx][3], poses[jx][4], poses[jx][5], poses[jx][6] };

    double tij[3], qij[4];
    relSE3(ti, qi, tj, qj, tij, qij);

    double Xi[4], Xj[4];
    for (int i=0; i<patches.size(2); i++) {
      for (int j=0; j<patches.size(3); j++) {
        
        Xi[0] = (patches[kx][0][i][j] - cx) / fx;
        Xi[1] = (patches[kx][1][i][j] - cy) / fy;
        Xi[2] = 1.0;
        Xi[3] = patches[kx][2][i][j];

        actSE3(tij, qij, Xi, Xj);

        coords[n][0][i][j] = fx * (Xj[0] / Xj[2]) + cx;
        coords[n][1][i][j] = fy * (Xj[1] / Xj[2]) + cy;
        // coords[n][2][i][j] = 1.0 / Xj[2];

      }
    }
  }
}


__global__ void pose_retr_kernel(const int t0, const int t1,
    torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> poses,
    torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> update)
{
  GPU_1D_KERNEL_LOOP(i, t1 - t0) {
    //printf("[LOG] pose_retr_cpu loop i = %d\n", i);

    const double t = t0 + i;
    double t1[3], t0[3] = { poses[t][0], poses[t][1], poses[t][2] };
    double q1[4], q0[4] = { poses[t][3], poses[t][4], poses[t][5], poses[t][6] };

    double xi[6] = {
      update[i][0],
      update[i][1],
      update[i][2],
      update[i][3],
      update[i][4],
      update[i][5],
    };

    retrSE3(xi, t0, q0, t1, q1);

    poses[t][0] = t1[0];
    poses[t][1] = t1[1];
    poses[t][2] = t1[2];
    poses[t][3] = q1[0];
    poses[t][4] = q1[1];
    poses[t][5] = q1[2];
    poses[t][6] = q1[3];
  }
}


__global__ void patch_retr_kernel(
    torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> index,
    torch::PackedTensorAccessor32<double,4,torch::RestrictPtrTraits> patches,
    torch::PackedTensorAccessor32<double,1,torch::RestrictPtrTraits> update)
{
  GPU_1D_KERNEL_LOOP(n, index.size(0)) {
    //printf("[LOG] patch_retr_cpu loop n = %d\n", n);
    const int p = patches.size(2);
    const int ix = index[n];
  
    double d = patches[ix][2][0][0];
    d = d + update[n];
    d = (d > 20) ? 1.0 : d;
    d = max(d, 1e-4);

    for (int i=0; i<p; i++) {
      for (int j=0; j<p; j++) {
        patches[ix][2][i][j] = d;
      }
    }
  }
  //printf("[LOG] patch_retr_cpu finished\n");

}


__global__ void reprojection_residuals_and_hessian_debug(
    const torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> poses,
    const torch::PackedTensorAccessor32<double,4,torch::RestrictPtrTraits> patches,
    const torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> intrinsics,
    const torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> target,
    const torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> weight,
    const torch::PackedTensorAccessor32<double,1,torch::RestrictPtrTraits> lmbda,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ii,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> jj,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> kk,
    const torch::PackedTensorAccessor32<long,1,torch::RestrictPtrTraits> ku,
    torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> B,
    torch::PackedTensorAccessor32<double,2,torch::RestrictPtrTraits> E,
    torch::PackedTensorAccessor32<double,1,torch::RestrictPtrTraits> C,
    torch::PackedTensorAccessor32<double,1,torch::RestrictPtrTraits> v,
    torch::PackedTensorAccessor32<double,1,torch::RestrictPtrTraits> u, const int t0)
{

  __shared__ double fx, fy, cx, cy;
  if (threadIdx.x == 0) {
    fx = intrinsics[0][0];
    fy = intrinsics[0][1];
    cx = intrinsics[0][2];
    cy = intrinsics[0][3];
  }

  __syncthreads();

  GPU_1D_KERNEL_LOOP(n, ii.size(0)) {
    int k = ku[n];
    int ix = ii[n];
    int jx = jj[n];
    int kx = kk[n];

    double ti[3] = { poses[ix][0], poses[ix][1], poses[ix][2] };
    double tj[3] = { poses[jx][0], poses[jx][1], poses[jx][2] };
    double qi[4] = { poses[ix][3], poses[ix][4], poses[ix][5], poses[ix][6] };
    double qj[4] = { poses[jx][3], poses[jx][4], poses[jx][5], poses[jx][6] };

    double Xi[4], Xj[4];
    Xi[0] = (patches[kx][0][1][1] - cx) / fx;
    Xi[1] = (patches[kx][1][1][1] - cy) / fy;
    Xi[2] = 1.0;
    Xi[3] = patches[kx][2][1][1];
    
    double tij[3], qij[4];
    relSE3(ti, qi, tj, qj, tij, qij);
    actSE3(tij, qij, Xi, Xj);

    const double X = Xj[0];
    const double Y = Xj[1];
    const double Z = Xj[2];
    const double W = Xj[3];

    const double d = (Z >= 0.2) ? 1.0 / Z : 0.0; 
    const double d2 = d * d;

    const double x1 = fx * (X / Z) + cx;
    const double y1 = fy * (Y / Z) + cy;

    const double rx = target[n][0] - x1;
    const double ry = target[n][1] - y1;

    const bool in_bounds = (sqrt(rx*rx + ry*ry) < 128) && (Z > 0.2) &&
      (x1 > -64) && (y1 > -64) && (x1 < 2*cx + 64) && (y1 < 2*cy + 64);

    const double mask = in_bounds ? 1.0 : 0.0;

    ix = ix - t0;
    jx = jx - t0;

    {
      const double r = target[n][0] - x1;
      const double w = mask * weight[n][0];

      double Jz = fx * (tij[0] * d - tij[2] * (X * d2));
      double Ji[6], Jj[6] = {fx*W*d, 0, fx*-X*W*d2, fx*-X*Y*d2, fx*(1+X*X*d2), fx*-Y*d};
      //printf("[LOG] Calculating adjSE3 for n = %d, ix = %d, jx = %d\n", n, ix, jx);

      adjSE3(tij, qij, Jj, Ji);
      //printf("[LOG] Finished adjSE3 for n = %d, ix = %d, jx = %d\n", n, ix, jx);

      //printf("[LOG] Updating B, E, C, v, u for n = %d, ix = %d, jx = %d\n", n, ix, jx);
      for (int i=0; i<6; i++) {
        for (int j=0; j<6; j++) {
          if (ix >= 0)
            atomicAdd(&B[6*ix+i][6*ix+j],  w * Ji[i] * Ji[j]);
          if (jx >= 0)
            atomicAdd(&B[6*jx+i][6*jx+j],  w * Jj[i] * Jj[j]);
          if (ix >= 0 && jx >= 0) {
            atomicAdd(&B[6*ix+i][6*jx+j], -w * Ji[i] * Jj[j]);
            atomicAdd(&B[6*jx+i][6*ix+j], -w * Jj[i] * Ji[j]);
          }
        }
      }
      //printf("[LOG] Finished updating B for n = %d, ix = %d, jx = %d\n", n, ix, jx);
      for (int i=0; i<6; i++) {
        if (ix >= 0)
          atomicAdd(&E[6*ix+i][k], -w * Jz * Ji[i]);
        if (jx >= 0)
          atomicAdd(&E[6*jx+i][k],  w * Jz * Jj[i]);
      }

      //printf("[LOG] Finished updating E for n = %d, ix = %d, jx = %d\n", n, ix, jx);
      for (int i=0; i<6; i++) {
        if (ix >= 0)
          atomicAdd(&v[6*ix+i], -w * r * Ji[i]);
        if (jx >= 0)
          atomicAdd(&v[6*jx+i],  w * r * Jj[i]);
      }

      atomicAdd(&C[k], w * Jz * Jz);
      atomicAdd(&u[k], w *  r * Jz);
    }

    {
      const double r = target[n][1] - y1;
      const double w = mask * weight[n][1];
      
      double Jz = fy * (tij[1] * d - tij[2] * (Y * d2));
      double Ji[6], Jj[6] = {0, fy*W*d, fy*-Y*W*d2, fy*(-1-Y*Y*d2), fy*(X*Y*d2), fy*X*d};


      //printf("[LOG] Calculating adjSE3 for n = %d, ix = %d, jx = %d\n", n, ix, jx);
      adjSE3(tij, qij, Jj, Ji);
      //printf("[LOG] Finished adjSE3 for n = %d, ix = %d, jx = %d\n", n, ix, jx);

      //printf("[LOG] Updating B, E, C, v, u for n = %d, ix = %d, jx = %d\n", n, ix, jx);
      for (int i=0; i<6; i++) {
        for (int j=0; j<6; j++) {
          if (ix >= 0)
            atomicAdd(&B[6*ix+i][6*ix+j],  w * Ji[i] * Ji[j]);
          if (jx >= 0)
            atomicAdd(&B[6*jx+i][6*jx+j],  w * Jj[i] * Jj[j]);
          if (ix >= 0 && jx >= 0) {
            atomicAdd(&B[6*ix+i][6*jx+j], -w * Ji[i] * Jj[j]);
            atomicAdd(&B[6*jx+i][6*ix+j], -w * Jj[i] * Ji[j]);
          }
        }
      }
      //printf("[LOG] Finished updating B for n = %d, ix = %d, jx = %d\n", n, ix, jx);
      for (int i=0; i<6; i++) {
        if (ix >= 0)
          atomicAdd(&E[6*ix+i][k], -w * Jz * Ji[i]);
        if (jx >= 0)
          atomicAdd(&E[6*jx+i][k],  w * Jz * Jj[i]);
      }
      //printf("[LOG] Finished updating E for n = %d, ix = %d, jx = %d\n", n, ix, jx);
      for (int i=0; i<6; i++) {
        if (ix >= 0)
          atomicAdd(&v[6*ix+i], -w * r * Ji[i]);
        if (jx >= 0)
          atomicAdd(&v[6*jx+i],  w * r * Jj[i]);
      }

      //printf("[LOG] Finished updating v for n = %d, ix = %d, jx = %d\n", n, ix, jx);
      atomicAdd(&C[k], w * Jz * Jz);
      atomicAdd(&u[k], w *  r * Jz);
    }
  }
}





//FORWARD FUNCTION IMPLEMENTING THE BUNDLE ADJUSTMENT
std::vector<torch::Tensor> cuda_ba_debug(
    torch::Tensor poses,
    torch::Tensor patches,
    torch::Tensor intrinsics,
    torch::Tensor target,
    torch::Tensor weight,
    torch::Tensor lmbda,
    torch::Tensor ii,
    torch::Tensor jj, 
    torch::Tensor kk,
    const int t0, const int t1, const int iterations)
{

  auto ktuple = torch::_unique(kk, true, true);
  torch::Tensor kx = std::get<0>(ktuple);
  torch::Tensor ku = std::get<1>(ktuple);

  const int N = t1 - t0;    // number of poses
  const int M = kx.size(0); // number of patches
  const int P = patches.size(3); // patch size

  auto opts = torch::TensorOptions()
    .dtype(torch::kDouble).device(torch::kCUDA);

  poses = poses.view({-1, 7});
  patches = patches.view({-1,3,P,P});
  intrinsics = intrinsics.view({-1, 4});

  target = target.view({-1, 2});
  weight = weight.view({-1, 2});

  const int num = ii.size(0);
  torch::Tensor B = torch::empty({6*N, 6*N}, opts);
  torch::Tensor E = torch::empty({6*N, 1*M}, opts);
  torch::Tensor C = torch::empty({M}, opts);

  torch::Tensor v = torch::empty({6*N}, opts);
  torch::Tensor u = torch::empty({1*M}, opts);

  for (int itr=0; itr < iterations; itr++) {

    B.zero_();
    E.zero_();
    C.zero_();
    v.zero_();
    u.zero_();

    v = v.view({6*N});
    u = u.view({1*M});

    //printf("[LOG] Starting reprojection_residuals_and_hessian kernel\n");
    reprojection_residuals_and_hessian_debug<<<NUM_BLOCKS(ii.size(0)), NUM_THREADS>>>(
      poses.packed_accessor32<double,2,torch::RestrictPtrTraits>(),
      patches.packed_accessor32<double,4,torch::RestrictPtrTraits>(),
      intrinsics.packed_accessor32<double,2,torch::RestrictPtrTraits>(),
      target.packed_accessor32<double,2,torch::RestrictPtrTraits>(),
      weight.packed_accessor32<double,2,torch::RestrictPtrTraits>(),
      lmbda.packed_accessor32<double,1,torch::RestrictPtrTraits>(),
      ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      kk.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      ku.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
      B.packed_accessor32<double,2,torch::RestrictPtrTraits>(),
      E.packed_accessor32<double,2,torch::RestrictPtrTraits>(),
      C.packed_accessor32<double,1,torch::RestrictPtrTraits>(),
      v.packed_accessor32<double,1,torch::RestrictPtrTraits>(),
      u.packed_accessor32<double,1,torch::RestrictPtrTraits>(), t0);

    v = v.view({6*N, 1});
    u = u.view({1*M, 1});

    //peinr log tensor stats fo the output tensors
    log_tensor_stats(B, "B");
    log_tensor_stats(E, "E");
    log_tensor_stats(C, "C");
    log_tensor_stats(v, "v");
    log_tensor_stats(u, "u");

    torch::Tensor Q = 1.0 / (C + lmbda).view({1, M});

    log_tensor_stats(Q, "Q");

    if (t1 - t0 == 0) {

      torch::Tensor Qt = torch::transpose(Q, 0, 1);
      torch::Tensor dZ = Qt * u;

      dZ = dZ.view({M});
      log_tensor_stats(dZ, "dZ");

      patch_retr_kernel<<<NUM_BLOCKS(M), NUM_THREADS>>>(
        kx.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
        patches.packed_accessor32<double,4,torch::RestrictPtrTraits>(),
        dZ.packed_accessor32<double,1,torch::RestrictPtrTraits>());

    }

    else {

      torch::Tensor EQ = E * Q;
      log_tensor_stats(EQ, "EQ");
      torch::Tensor Et = torch::transpose(E, 0, 1);
      torch::Tensor Qt = torch::transpose(Q, 0, 1);


      torch::Tensor S = B - torch::matmul(EQ, Et);
      
      torch::Tensor y = v - torch::matmul(EQ,  u);
      log_tensor_stats(y, "y");

      torch::Tensor I = torch::eye(6*N, opts);
      S += I * (1e-4 * S + 1.0);
      log_tensor_stats(S, "S");

      //printf("[LOG] S shape: ");
      torch::Tensor U = torch::linalg::cholesky(S);
      log_tensor_stats(U, "U");
      torch::Tensor dX = torch::cholesky_solve(y, U);
      log_tensor_stats(dX, "dX");
      torch::Tensor dZ = Qt * (u - torch::matmul(Et, dX));
      log_tensor_stats(dZ, "dZ");

      dX = dX.view({N, 6});
      dZ = dZ.view({M});
      //printf("[LOG] pose retr_kernel with t0 = %d, t1 = %d\n", t0, t1);

      pose_retr_kernel<<<NUM_BLOCKS(N), NUM_THREADS>>>(t0, t1,
          poses.packed_accessor32<double,2,torch::RestrictPtrTraits>(),
          dX.packed_accessor32<double,2,torch::RestrictPtrTraits>());

      //printf("[LOG] patch retr_kernel with M = %d\n", M);
      patch_retr_kernel<<<NUM_BLOCKS(M), NUM_THREADS>>>(
          kx.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
          patches.packed_accessor32<double,4,torch::RestrictPtrTraits>(),
          dZ.packed_accessor32<double,1,torch::RestrictPtrTraits>());
      
      //printf("[LOG] Finished patch retr_kernel with M = %d\n", M);

    }
  }
  //printf("[LOG] Finished all iterations in cuda_ba_log\n");

  return {};
}


torch::Tensor cuda_reproject(
    torch::Tensor poses,
    torch::Tensor patches,
    torch::Tensor intrinsics,
    torch::Tensor ii,
    torch::Tensor jj, 
    torch::Tensor kk)
{

  const int N = ii.size(0);
  const int P = patches.size(3); // patch size

  poses = poses.view({-1, 7});
  patches = patches.view({-1,3,P,P});
  intrinsics = intrinsics.view({-1, 4});

  auto opts = torch::TensorOptions()
    .dtype(torch::kDouble).device(torch::kCUDA);

  torch::Tensor coords = torch::empty({N, 2, P, P}, opts);

  reproject<<<NUM_BLOCKS(N), NUM_THREADS>>>(
    poses.packed_accessor32<double,2,torch::RestrictPtrTraits>(),
    patches.packed_accessor32<double,4,torch::RestrictPtrTraits>(),
    intrinsics.packed_accessor32<double,2,torch::RestrictPtrTraits>(),
    ii.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    jj.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    kk.packed_accessor32<long,1,torch::RestrictPtrTraits>(),
    coords.packed_accessor32<double,4,torch::RestrictPtrTraits>());

  return coords.view({1, N, 2, P, P});

}